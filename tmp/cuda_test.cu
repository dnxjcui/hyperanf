
#include <hip/hip_runtime.h>
#include <iostream>
#include <math.h>

using namespace std;


__global__
void add(int n, float *x, float *y){
    int index = threadIdx.x;
    int stride = blockDim.x;
    for (int i = index; i < n; i += stride){
        y[i] = x[i] + y[i];
    }
}

int main(){    
    int N = 1 << 24;
    float *x, *y;
    hipMallocManaged(&x, N * sizeof(float));
    hipMallocManaged(&y, N * sizeof(float));

    for (int i = 0; i < N; i++){
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    int blockSize = 1024;
    int gridSize = 12;

    add<<<gridSize, blockSize>>>(N, x, y);

    hipDeviceSynchronize();

    float maxError = 0.0f;
    for (int i = 0; i < N; i++){
        maxError = fmax(maxError, fabs(y[i] - 3.0f));
    }
    std::cout << "Max error: " << maxError << std::endl;

    hipFree(x);
    hipFree(y);
    return 0;
}